#include "hip/hip_runtime.h"
//
// Created by Leung See Ho on 14/10/2023.
//

#include "kernel.cuh"
#include <algorithm>
#include <cmath>

// declare function to be ran in the GPU
__global__ void monte_carlo_sim(double* d_St, const double* d_Zs, double d_drift, double d_sigma,
                                double d_deltaT, int d_n, int d_m) {

    // create stock_index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // for each simluated stock
    if (idx < d_m) {
        int t = 0;
        while (t < d_n) {
            // compute stock price from previous stock price
            d_St[idx] = d_St[idx] * d_drift * exp((d_sigma) * sqrt(d_deltaT) * d_Zs[idx*(d_n) + t]);
            t++;
        }
    }
}

// function to setup running Monte Carlo Simulation in the GPU
void run_mc_cuda(double* ST, double S, double sigma, double rate, double T, int n,
                 int m, double* Zs) {

    // assign values in the CPU
    double deltaT = T/n;
    double drift = exp((rate - 0.5*(pow(sigma, 2.0)))*T/n);
    for (size_t i=0; i<m; i++) ST[i] = S;

    // declarations for variables in the GPU
    //array
    double* d_St = nullptr;
    double* d_Zs = nullptr;

    // allocations in GPU (device) global memory
    hipMalloc((void **) &d_Zs, n*m*sizeof(double));
    hipMalloc((void **) &d_St, m*sizeof(double));
    // copy values from host (CPU) to device (GPU)
    hipMemcpy(d_Zs, Zs, n*m*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_St, ST, m*sizeof(double), hipMemcpyHostToDevice);

    // Use 1024 for THREADS_PER_BLOCK
    int TPB = 1024;

    // invoke the Monte Carlo kernel
    monte_carlo_sim<<<(m+TPB-1)/TPB,TPB>>>(d_St, d_Zs, drift, sigma, deltaT, n, m);

    // copy the results from the device (GPU) back to the host (CPU)
    hipMemcpy(ST, d_St, m*sizeof(double), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_Zs);
    hipFree(d_St);
}

// function to setup running Monte Carlo Simulation in the GPU
void run_mc_cuda(double* ST, double S, double sigma, double rate, double T, int n,
                 int m) {

    // assign values in the CPU
    double deltaT = T/n;
    double drift = exp((rate - 0.5*(pow(sigma, 2.0)))*T/n);
    for (size_t i=0; i<m; i++) ST[i] = S;

    // declarations for variables in the GPU
    //array
    double* d_St = nullptr;
    double* d_Zs = nullptr;

    // allocations in GPU (device) global memory
    hipMalloc((void **) &d_Zs, n*m*sizeof(double));
    hipMalloc((void **) &d_St, m*sizeof(double));
    // copy values from host (CPU) to device (GPU)
    hipMemcpy(d_Zs, Zs, n*m*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_St, ST, m*sizeof(double), hipMemcpyHostToDevice);

    // Use 1024 for THREADS_PER_BLOCK
    int TPB = 1024;

    // invoke the Monte Carlo kernel
    monte_carlo_sim<<<(m+TPB-1)/TPB,TPB>>>(d_St, d_Zs, drift, sigma, deltaT, n, m);

    // copy the results from the device (GPU) back to the host (CPU)
    hipMemcpy(ST, d_St, m*sizeof(double), hipMemcpyDeviceToHost);

    // free device memory
    hipFree(d_Zs);
    hipFree(d_St);
}